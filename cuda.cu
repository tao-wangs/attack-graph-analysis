#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>

__global__ void myKernel(int node) {
    // Example kernel work
}

// Launch n amount of kernels in parallel
for (int i = 0; i < numNodes; ++i) {
    if (dependencyCounts[i] == 0) {
        myKernel<<<1, 256, 0, streams[i]>>>(i);
        hipEventRecord(events[i], streams[i]);
    }
}

int main() {
    // Example DAG represented with dependency counts and adjacency list
    std::vector<int> dependencyCounts = {0, 1, 1, 2}; // Number of dependencies for each node
    std::vector<std::vector<int>> dependents = {{1, 2}, {}, {}, {}}; // Nodes dependent on each node

    int numNodes = dependencyCounts.size();
    std::vector<hipStream_t> streams(numNodes);
    std::vector<hipEvent_t> events(numNodes);

    // Initialize streams and events
    for (int i = 0; i < numNodes; ++i) {
        hipStreamCreate(&streams[i]);
        hipEventCreate(&events[i]);
    }

    // Launch initial nodes with zero dependencies
    for (int i = 0; i < numNodes; ++i) {
        if (dependencyCounts[i] == 0) {
            myKernel<<<1, 256, 0, streams[i]>>>(i);
            hipEventRecord(events[i], streams[i]);
        }
    }

    // Process nodes as their dependencies are resolved
    for (int i = 0; i < numNodes; ++i) {
        for (int dep : dependents[i]) {
            // Wait on the event in the dependent node's stream
            hipStreamWaitEvent(streams[dep], events[i], 0);
            if (--dependencyCounts[dep] == 0) {
                // Dependencies resolved, launch the kernel
                myKernel<<<1, 256, 0, streams[dep]>>>(dep);
                hipEventRecord(events[dep], streams[dep]);
            }
        }
    }

    // Synchronize all streams at the end
    for (int i = 0; i < numNodes; ++i) {
        hipStreamSynchronize(streams[i]);
    }

    // Cleanup
    for (int i = 0; i < numNodes; ++i) {
        hipStreamDestroy(streams[i]);
        hipEventDestroy(events[i]);
    }

    return 0;
}


// Assume dependencyCounts and dependents are initialized appropriately
while (not all nodes processed) {
    for each node in DAG {
        if (node's dependencies are resolved and not yet processed) {
            Launch kernel for node in its stream
            Record event for node's completion
            Mark node as processed
        }
    }
    for each node in DAG {
        if (node's kernel was launched) {
            Wait for node's event to complete
            Update dependency counts of node's dependents
        }
    }
}

